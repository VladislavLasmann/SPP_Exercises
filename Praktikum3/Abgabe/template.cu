#include "hip/hip_runtime.h"
/**
 * Gruppe: 122
 * Konstantin Müller (2327697) 
 * Robin Ferrari 	 (2585277) 
 * Vladislav Lasmann (2593078)
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <math.h>
 #include <string.h>
 #include <time.h>
 #include <sys/time.h>
 #include "timing.h"
 
 #define BLOCK_SIZE  16
 #define HEADER_SIZE 122
 
 typedef unsigned char BYTE;
 
 //#define CUDA_ERROR_CHECK
 
 #define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
 #define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
 
 void showGPUMem();
 
 inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
 {
 #ifdef CUDA_ERROR_CHECK
     if ( hipSuccess != err )
     {
         fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
         exit( -1 );
     }
 #endif
     
     return;
 }
 
 inline void __cudaCheckError( const char *file, const int line )
 {
 #ifdef CUDA_ERROR_CHECK
     hipError_t err = hipGetLastError();
     if ( hipSuccess != err )
     {
         fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
     showGPUMem();
         exit( -1 );
     }
     
     // More careful checking. However, this will affect performance.
     // Comment away if needed.
     /*   err = hipDeviceSynchronize();
      if( hipSuccess != err )
      {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
      file, line, hipGetErrorString( err ) );
      exit( -1 );
      }*/
 #endif
     
     return;
 }
 
 void showGPUMem(){
     // show memory usage of GPU
     
     size_t free_byte ;
     
     size_t total_byte ;
     
     hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
     
     if ( hipSuccess != cuda_status ){
         
         printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
         
         exit(1);
         
     }
     
     
     
     double free_db = (double)free_byte ;
     
     double total_db = (double)total_byte ;
     
     double used_db = total_db - free_db ;
     
     printf("GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
 }
 
 
 /********* BMP Image functions **************/
 typedef struct
 {
     int   width;
     int   height;
     float *data;
 } BMPImage;
 
 BYTE bmp_info[HEADER_SIZE]; // Reference header
 
 
 /**
  * Reads a BMP 24bpp file and returns a BMPImage structure.
  * Thanks to https://stackoverflow.com/a/9296467
  */
 BMPImage readBMP(char *filename)
 {
     BMPImage bitmap = { 0 };
     int      size   = 0;
     BYTE     *data  = NULL;
     FILE     *file  = fopen(filename, "rb");
     
     // Read the header (expected BGR - 24bpp)
     fread(bmp_info, sizeof(BYTE), HEADER_SIZE, file);
     
     // Get the image width / height from the header
     bitmap.width  = *((int *)&bmp_info[18]);
     bitmap.height = *((int *)&bmp_info[22]);
     size          = *((int *)&bmp_info[34]);
     
     // Read the image data
     data = (BYTE *)malloc(sizeof(BYTE) * size);
     fread(data, sizeof(BYTE), size, file);
     
     // Convert the pixel values to float
     bitmap.data = (float *)malloc(sizeof(float) * size);
     
     for (int i = 0; i < size; i++)
     {
         bitmap.data[i] = (float)data[i];
     }
     
     fclose(file);
     free(data);
     
     return bitmap;
 }
 
 
 
 /**
  * Writes a BMP file in grayscale given its image data and a filename.
  */
 void writeBMPGrayscale(int width, int height, float *image, char *filename)
 {
     FILE *file = NULL;
     
     file = fopen(filename, "wb");
     
     // Write the reference header
     fwrite(bmp_info, sizeof(BYTE), HEADER_SIZE, file);
     
     // Unwrap the 8-bit grayscale into a 24bpp (for simplicity)
     for (int h = 0; h < height; h++)
     {
         int row = h * width;
         
         for (int w = 0; w < width; w++)
         {
             BYTE pixel = (BYTE)((image[row + w] > 255.0f) ? 255.0f :
                                 (image[row + w] < 0.0f)   ? 0.0f   :
                                 image[row + w]);
             
             // Repeat the same pixel value for BGR
             fputc(pixel, file);
             fputc(pixel, file);
             fputc(pixel, file);
         }
     }
     
     fclose(file);
 }
 
 /**
 * Releases a given BMPImage.
 */
 void freeBMP(BMPImage bitmap)
 {
     free(bitmap.data);
 }
 
 
 /*********** Gray Scale Filter  *********/
 
 /**
  * Converts a given 24bpp image into 8bpp grayscale using the CPU.
  */
 void grayscale(int width, int height, float *image, float *image_out)
 {
     for (int h = 0; h < height; h++)
     {
         int offset_out = h * width;      // 1 color per pixel
         int offset     = offset_out * 3; // 3 colors per pixel
         
         for (int w = 0; w < width; w++)
         {
             float *pixel = &image[offset + w * 3];
             
             // Convert to grayscale following the "luminance" model
             image_out[offset_out + w] = pixel[0] * 0.0722f + // B
             pixel[1] * 0.7152f + // G
             pixel[2] * 0.2126f;  // R
         }
     }
 }
 
 /**
  * Converts a given 24bpp image into 8bpp grayscale using the GPU.
  */
 __global__
 void cuda_grayscale(int width, int height, float *image, float *image_out)
 {   
     // calculate the global indices
     int absX = blockIdx.x * blockDim.x + threadIdx.x;
     int absY = blockIdx.y * blockDim.y + threadIdx.y;
     // calculate the offset in the memories
     int offset_out = absY * width + absX;   // 1 color per pixel
     int offset_in  = offset_out * 3;        // 3 color per pixel
 
     // Grayscaling from 3 colors per pixel to 1 color per pixel
     image_out[offset_out] = 0.0722f * image[ offset_in] +     // B
                             0.7152f * image[ offset_in + 1] + // G
                             0.2126f * image[ offset_in + 2];  // R
 }
 
 /****************Convolution Filters*****/
 
 
 /**
  * Applies a 3x3 convolution matrix to a pixel using the CPU.
  */
 float applyFilter(float *image, int stride, float *matrix, int filter_dim)
 {
     float pixel = 0.0f;
     
     for (int h = 0; h < filter_dim; h++)
     {
         int offset        = h * stride;
         int offset_kernel = h * filter_dim;
         
         for (int w = 0; w < filter_dim; w++)
         {
             pixel += image[offset + w] * matrix[offset_kernel + w];
         }
     }
     
     return pixel;
 }
 
 /**
  * Task 5
  * Applies a 3x3 convolution matrix to a pixel using the GPU.
  */
 __device__
 float cuda_applyFilter(float *image, int stride, float *matrix, int filter_dim)
 {
     float pixel = 0.0f;
     
     for (int h = 0; h < filter_dim; h++)
     {
         int offset        = h * stride;
         int offset_kernel = h * filter_dim;
         
         for (int w = 0; w < filter_dim; w++)
         {
             pixel += image[offset + w] * matrix[offset_kernel + w];
         }
     }
     
     return pixel;
 }
 
 /**
  * Applies a Gaussian 3x3 filter to a given image using the CPU.
  */
 void gaussian(int width, int height, float *image, float *image_out)
 {
     float gaussian[9] = {   1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                             2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                             1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };
     
     for (int h = 0; h < (height - 2); h++)
     {
         int offset_t = h * width;
         int offset   = (h + 1) * width;
         
         for (int w = 0; w < (width - 2); w++)
         {
             image_out[offset + (w + 1)] = applyFilter(&image[offset_t + w], width, gaussian, 3);
         }
     }
 }
 
 /**
  * Task 6
  * Applies a Gaussian 3x3 filter to a given image using the GPU.
  */
 __global__
 void cuda_gaussian(int width, int height, float *image, float *image_out)
 {
     //(9 pt): implement gaussian filter kernel
     float gaussian[9] = {   1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                             2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                             1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };
     
     // calculate the global indices
     int absX = blockIdx.x * blockDim.x + threadIdx.x;
     int absY = blockIdx.y * blockDim.y + threadIdx.y;
     
     // calculate the offset in the memory
     int offset_t    = absY * width;
     int offset      = (absY + 1) * width;
 
     if( absY < (height - 2) && absX < (width - 2) ){
         image_out[offset + (absX + 1)] = cuda_applyFilter(&image[offset_t + absX], width, gaussian, 3);
     }
 
 }
 
 /**
  * Calculates the gradient of an image using a Sobel filter on the CPU.
  */
 void sobel(int width, int height, float *image, float *image_out)
 {
     float sobel_x[9] = {1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
     float sobel_y[9] = {1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                         -1.0f, -2.0f, -1.0f };
     
     for (int h = 0; h < (height - 2); h++)
     {
         int offset_t = h * width;
         int offset   = (h + 1) * width;
         
         for (int w = 0; w < (width - 2); w++)
         {
             float gx = applyFilter(&image[offset_t + w], width, sobel_x, 3);
             float gy = applyFilter(&image[offset_t + w], width, sobel_y, 3);
             
             image_out[offset + (w + 1)] = sqrtf(gx * gx + gy * gy);
         }
     }
 }
 
 /**
  * Task 7
  * Calculates the gradient of an image using a Sobel filter on the GPU.
  */
 __global__
 void cuda_sobel(int width, int height, float *image, float *image_out)
 {
     //TODO (9 pt): implement sobel filter kernel
     float sobel_x[9] = {1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
     float sobel_y[9] = {1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                         -1.0f, -2.0f, -1.0f };
     
     // calculate the global indices
     int absX = blockIdx.x * blockDim.x + threadIdx.x;
     int absY = blockIdx.y * blockDim.y + threadIdx.y;
     
     // calculate the offset in the memory
     int offset_t    = absY * width;
     int offset      = (absY + 1) * width;
 
     if( absY < (height - 2) && absX < (width - 2) ){
         float gx = cuda_applyFilter(&image[offset_t + absX], width, sobel_x, 3);
         float gy = cuda_applyFilter(&image[offset_t + absX], width, sobel_y, 3);
 
         image_out[offset + (absX + 1)] = sqrtf(gx * gx + gy * gy);
     }
     
 }
 
 
 int main(int argc, char *argv[])
 {
     //check for arguments
     if(argc != 2){
         puts("Usage: sol input_file\n");
         exit(0);
     }
     
     BMPImage bitmap          = { 0 };
     float    *d_bitmap       = { 0 };
     float    *image_out[2]   = { 0 };
     float    *d_image_out[2] = { 0 };
     int      image_size      = 0;
     double   t[2]            = { 0 };
     //dim3     grid(1);
     dim3     block(BLOCK_SIZE, BLOCK_SIZE);
     char     path[255];
     
 
     init_clock_time();
     
     // Read the input image and update the grid dimension
     bitmap     = readBMP(argv[1]);
     image_size = bitmap.width * bitmap.height;
     
     /* Task 1: Calculate the grid dimensions-size
     */
     dim3 grid( ceil((double) bitmap.width / (double) BLOCK_SIZE), ceil( (double) bitmap.height / (double) BLOCK_SIZE) );
     
     printf("Image read (width=%d height=%d).\n", bitmap.width, bitmap.height);
     
     // Allocate the intermediate image buffers for each step
     for (int i = 0; i < 2; i++)
     {
         image_out[i] = (float *)calloc(image_size, sizeof(float));
         
         // Task2:
         //(2 pt): allocate memory on the device
         //(2 pt): intialize allocated memory on device to zero
         hipMalloc((void**) &d_image_out[i], image_size * sizeof(float) );
         hipMemcpy( (void*) d_image_out[i], image_out[i], image_size * sizeof(float), hipMemcpyHostToDevice );
     }
 
     //Task3: copy input image to device
     //(2 pt): Allocate memory on device for input image
     //(2 pt): Copy input image into the device memory
     hipMalloc( (void**) &d_bitmap, 3 * image_size * sizeof(float) );
     hipMemcpy( (void*) d_bitmap, bitmap.data, 3 * image_size * sizeof(float), hipMemcpyHostToDevice );
 
     t[0] = get_clock_time();
     
     // Covert input image to grayscale
     //grayscale(bitmap.width, bitmap.height, bitmap.data, image_out[0]); //serial version
     
     //Task4: (2 pt): Launch cuda_grayscale()
     cuda_grayscale<<<grid, block>>>(bitmap.width, bitmap.height, d_bitmap, d_image_out[0]);
 
     t[1] = get_clock_time();
 
     //(2 pt): transfer image from device to the main memory for saving onto the disk
     hipMemcpy( image_out[0], d_image_out[0], image_size * sizeof(float), hipMemcpyDeviceToHost );
     
     sprintf(path, "images/grayscale.bmp");
     writeBMPGrayscale(bitmap.width, bitmap.height, image_out[0], path); //write output file
     printf("Time taken for grayscaling: %8.5f ms\n",t[1] - t[0]);
     
     // Apply a 3x3 Gaussian filter
     t[0] = get_clock_time();
     // Launch the CPU version
     // gaussian(bitmap.width, bitmap.height, image_out[0], image_out[1]);
     
     // Launch the GPU version
     // (2 pt): Launch cuda_gaussian();
     cuda_gaussian<<<grid, block>>>(bitmap.width, bitmap.height, d_image_out[0], d_image_out[1]);
     
     
     t[1] = get_clock_time();
     //(2 pt): transfer image from device to the main memory for saving onto the disk
     hipMemcpy( image_out[1], d_image_out[1], image_size * sizeof(float), hipMemcpyDeviceToHost);
 
     // Store the result image with the Gaussian filter applied
     sprintf(path, "images/gaussian.bmp");
     writeBMPGrayscale(bitmap.width, bitmap.height, image_out[1], path); //write output file
     printf("Time taken for Gaussian filtering: %8.5f ms\n",t[1] - t[0]);
     
 
     // Apply a Sobel filter
     t[0] = get_clock_time();
     // Launch the CPU version
     //sobel(bitmap.width, bitmap.height, image_out[1], image_out[0]);
     
     // Launch the GPU version
     // (2 pt): Launch cuda_sobel();
     cuda_sobel<<<grid, block>>>(bitmap.width, bitmap.height, d_image_out[1], d_image_out[0]);
    
     t[1] = get_clock_time();
     //(2 pt): transfer image from device to the main memory for saving onto the disk
     hipMemcpy( image_out[0], d_image_out[0], image_size * sizeof(float), hipMemcpyDeviceToHost);
     
     // Store the final result image with the Sobel filter applied
     sprintf(path, "images/sobel.bmp");
     writeBMPGrayscale(bitmap.width, bitmap.height, image_out[0], path); //write output file
     printf("Time taken for Sobel filtering: %8.5f ms\n",t[1] - t[0]);
     
     
     // Task 9
     // Release the allocated memory
     for (int i = 0; i < 2; i++)
     {
         free(image_out[i]);
         hipFree(d_image_out[i]);
     }
     
     freeBMP(bitmap);
     hipFree(d_bitmap);
     
     // Got any errors?
     printf("%s\n", hipGetErrorString(hipGetLastError()));
     return 0;
 }
 